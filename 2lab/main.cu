#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <time.h>
#include <math.h>

#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat = value;\
	if (_m_cudaStat != hipSuccess) {\
	 fprintf(stderr, "Error %s at line %d in file %s\n",\
	 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
	 exit(1);\
	} }

double w_time()
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + tv.tv_usec * 1E-6;
}

__global__ void init(float *a, float *b)
{
	a[threadIdx.x + blockDim.x * blockIdx.x] = threadIdx.x + blockDim.x * blockIdx.x;
	b[threadIdx.x + blockDim.x * blockIdx.x] = threadIdx.x + blockDim.x * blockIdx.x;
}

__global__ void compute(float *a, float *b, float *c)
{
	c[threadIdx.x + blockDim.x * blockIdx.x] = a[threadIdx.x + blockDim.x * blockIdx.x] + b[threadIdx.x + blockDim.x * blockIdx.x];
}

int main()
{
	int blocks = 781;
	int th_p_block = 128;
	int N = blocks * th_p_block;
	float *a_device;
	float *b_device;
	float *c_device;
	float *buffer_host;
	float elapsedTime;
	int start = pow(2, 10);
	int end = pow(2, 15); // in the task 2^23 
	hipEvent_t startEvent;
	hipEvent_t stopEvent;

	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	while (start < end) {
		printf("start = %d\n", start);
		blocks = start;
		th_p_block = 1;
		N = blocks * th_p_block;
		
		while (th_p_block <= 32) {
			// printf("N = %d\tblocks = %d\tth_p_block = %d\n", N, blocks, th_p_block);
	
			// elapsedTime = 0;
	
			buffer_host = (float *)malloc(N * sizeof(float));
	
			CUDA_CHECK_RETURN(hipMalloc(&a_device, N * sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc(&b_device, N * sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc(&c_device, N * sizeof(float)));
	
			init<<<blocks, th_p_block>>>(a_device, b_device);
	
			// elapsedTime -= w_time();
			hipEventRecord(startEvent, 0);

			compute<<<blocks, th_p_block>>>(a_device, b_device, c_device);

			hipEventRecord(stopEvent, 0);
			hipEventSynchronize(stopEvent);

			hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);

			// CUDA_CHECK_RETURN(cudaDeviceSynchronize());
			CUDA_CHECK_RETURN(hipGetLastError()); 
			// elapsedTime += w_time();
	
			printf("%f\n", elapsedTime);
	
			CUDA_CHECK_RETURN(hipMemcpy(buffer_host, c_device, N * sizeof(float), hipMemcpyDeviceToHost));
	
			free(buffer_host);
			hipFree(a_device);
			hipFree(b_device);
			hipFree(c_device);
			
			blocks /= 2;
			th_p_block *= 2;
			N = blocks * th_p_block;
		}

		start += 32;

		printf("\n");
	}
		
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	return 0;
}